#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <assert.h>
#include <hip/hip_runtime.h>
// #include "cutil.h"

#include "mex.h"

#define SPXL 8
#define SPXR 7
#define SPYU 8
#define SPYD 7

#define BLOCK_SIZE_Y 16   //SPYU+SPYD+1
#define BLOCK_SIZE_X 16		//SPYU+SPYD+1
#define MAX_CELL_SIZE 50

#define min(a, b) ((a) < (b) ? (a) : (b))
#define max(a, b) ((a) > (b) ? (a) : (b))

#define  MAX_LEVELS  1000

int quickSort(float *, int, int *, int *);
__global__ void NSSD(float *, float *, int , int , int *, int *, int *, int *, float *, int *, int *, float *, float *);


int quickSort(float *arr, int elements, int *arr1, int *arr2) 
{
	float  piv, piv1;
	int p1, p2;
	int beg[MAX_LEVELS], end[MAX_LEVELS], i=0, L, R ;
	beg[0]=0; end[0]=elements;

	while (i>=0) {
		L=beg[i]; R=end[i]-1;
		if (L<R) {
			piv=(arr[L]); 
			piv1=arr[L]; 
			p1=arr1[L]; 
			p2=arr2[L]; 
			if (i==MAX_LEVELS-1) 
				return 0;
			while (L<R) {
				while ((arr[R])>=piv && L<R)
					R--; 
				if (L<R) {
					arr[L]=arr[R];
					arr1[L]=arr1[R];
					arr2[L++]=arr2[R];
				}
				while ((arr[L])<=piv && L<R) 
					L++; 
				if (L<R) {
					arr[R]=arr[L]; 
					arr1[R]=arr1[L];
					arr2[R--]=arr2[L];
				}

			}
			arr[L]=piv1; 
			arr1[L]=p1; 
			arr2[L]=p2; 
			beg[i+1]=L+1; 
			end[i+1]=end[i]; 
			end[i++]=L; 
		}
		else 
			i--; 
	}
	return 1; 
}

__global__ void NSSD(float *Im1, float *Im2, int IMSIZEX, int IMSIZEY, int *xmin, int *xmax, int *ymin, int *ymax, float *C, int *movedE, int *movedS, float *Sp, float *SpVal)
{
	// Block index
	int bID = blockIdx.x;

	
	// Thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int tid = ty*BLOCK_SIZE_X+tx;

	int tSizeX, tSizeY, tStartX, tStartY;
	int x, y;
	int saStartX, saStartY, saEndX, saEndY, saSizeX, saSizeY;


	tStartX = xmin[bID]-1;
	tStartY = ymin[bID]-1;
	tSizeX = xmax[bID] - tStartX;
	tSizeY = ymax[bID] - tStartY;
	

	saStartX = max(0, tStartX - SPXL);
	saStartY = max(0, tStartY - SPYU);
	saEndX = min(IMSIZEX-1, xmax[bID]-1 + SPXR);
	saEndY = min(IMSIZEY-1, ymax[bID]-1 + SPYD);
	saSizeX = saEndX - saStartX +1;
	saSizeY = saEndY - saStartY +1;

	__shared__ char SP[MAX_CELL_SIZE][MAX_CELL_SIZE];
	__shared__ float T[MAX_CELL_SIZE][MAX_CELL_SIZE];

	__shared__ float meanT;
	__shared__ float SumToffsetSqrd ;
	
	meanT = 0.;
	SumToffsetSqrd = 0.;

	__shared__ float temp[BLOCK_SIZE_X*BLOCK_SIZE_Y];
	__shared__ int tempi[BLOCK_SIZE_X*BLOCK_SIZE_Y];

	float meanF = 0.;
	float SumFoffsetSqrd = 0.;
	float numerator = 0.;

	__shared__ int noOfOnPixels;

	__shared__ int noOfPixelsToWriteInCellX;
	__shared__ int noOfPixelsToWriteInCellY;
	//__shared__ int noOfPixelsToWriteInSAX;
	//__shared__ int noOfPixelsToWriteInSAY;

	noOfPixelsToWriteInCellX = int((1.*tSizeX/BLOCK_SIZE_X)+1);
	noOfPixelsToWriteInCellY = int((1.*tSizeY/BLOCK_SIZE_X)+1);
	//noOfPixelsToWriteInSAX = int((1.*saSizeX/BLOCK_SIZE_X)+1);
	//noOfPixelsToWriteInSAY = int((1.*saSizeY/BLOCK_SIZE_X)+1);

	tempi[tid] = 0;
	for (x = noOfPixelsToWriteInCellX*tx ; x<min(noOfPixelsToWriteInCellX*(tx+1), tSizeX); x++ )
		for (y = noOfPixelsToWriteInCellY*ty ; y<min(noOfPixelsToWriteInCellY*(ty+1), tSizeY);  y++  )
		{
			T[x][y] = Im1[(tStartX +x)*IMSIZEY + tStartY + y];
			SP[x][y] = char(Sp[(tStartX +x)*IMSIZEY + tStartY + y]== SpVal[bID]);
			tempi[tid]+=int(SP[x][y]);
		}


    for(int stride = BLOCK_SIZE_X*BLOCK_SIZE_Y / 2; stride > 0; stride >>= 1)
	{
		__syncthreads();
        for(int iAccum = tid; iAccum < stride; iAccum += BLOCK_SIZE_X*BLOCK_SIZE_Y)
			tempi[iAccum] += tempi[stride + iAccum];
	}


	noOfOnPixels = tempi[0];
	__syncthreads();

	
	temp[tid] = 0.;
	for (x = noOfPixelsToWriteInCellX*tx ; x<min(noOfPixelsToWriteInCellX*(tx+1), tSizeX); x++ )
		for (y = noOfPixelsToWriteInCellY*ty ; y<min(noOfPixelsToWriteInCellY*(ty+1), tSizeY); y++ )
			if (SP[x][y])
				temp[tid] += T[x][y]/noOfOnPixels;


    for(int stride = BLOCK_SIZE_X*BLOCK_SIZE_Y / 2; stride > 0; stride >>= 1)
	{
		__syncthreads();
        for(int iAccum = tid; iAccum < stride; iAccum += BLOCK_SIZE_X*BLOCK_SIZE_Y)
			temp[iAccum] += temp[stride + iAccum];
	}

	meanT = temp[0];
	__syncthreads();



	temp[tid] =0.;
	for (x = noOfPixelsToWriteInCellX*tx ; x<min(noOfPixelsToWriteInCellX*(tx+1), tSizeX); x++ )
		for (y = noOfPixelsToWriteInCellY*ty ; y<min(noOfPixelsToWriteInCellY*(ty+1), tSizeY); y++ )
			if (SP[x][y])
				temp[tid] += pow(T[x][y] - meanT,2);
   
	for(int stride = BLOCK_SIZE_X*BLOCK_SIZE_Y / 2; stride > 0; stride >>= 1)
	{
		__syncthreads();
        for(int iAccum = tid; iAccum < stride; iAccum += BLOCK_SIZE_X*BLOCK_SIZE_Y)
			temp[iAccum] += temp[stride + iAccum];
	}

	SumToffsetSqrd = temp[0];
	__syncthreads();


	//for (x = noOfPixelsToWriteInSAX*tx ; x<min(noOfPixelsToWriteInSAX*(tx+1), saSizeX); x++ )
	//	for (y = noOfPixelsToWriteInSAY*ty ; y<min(noOfPixelsToWriteInSAY*(ty+1), saSizeY); y++ )	{
	//			SA[x][y] = Im2[(saStartX +x)*IMSIZEY + saStartY + y];
	//	}
	//__syncthreads();

	// Final x,y positions in search area where each threads writes an element
	int sax2 = min(tx + tSizeX -1, saSizeX-1);
	int say2 = min(ty + tSizeY -1, saSizeY-1);

	meanF =0.;
	for (x=tx; x<=sax2 ; x++)
		for (y=ty; y<=say2; y++)
			if (SP[x-tx][y-ty])
				meanF += Im2[(saStartX +x)*IMSIZEY + saStartY + y] / noOfOnPixels;


	__syncthreads();

	SumFoffsetSqrd =0.;
	for (x=tx; x<=sax2 ; x++)
		for (y=ty; y<=say2; y++)
			if (SP[x-tx][y-ty])
				SumFoffsetSqrd += pow(Im2[(saStartX +x)*IMSIZEY + saStartY + y] - meanF, 2);
	__syncthreads();


	float denom = SumToffsetSqrd + SumFoffsetSqrd;

	for (x=tx; x<=sax2 ; x++)
		for (y=ty; y<=say2; y++)
			if (SP[x-tx][y-ty])
				numerator += pow(T[x-tx][y-ty] - meanT - Im2[(saStartX +x)*IMSIZEY + saStartY + y] + meanF, 2);

	numerator = float(0.5) * numerator;
	__syncthreads();


	C[bID * BLOCK_SIZE_X * BLOCK_SIZE_Y + BLOCK_SIZE_X * ty + tx] =  -10.5 * (numerator/denom - 0.3) * (tx + tSizeX -1 < saSizeX) * (ty + tSizeY -1 < saSizeY);
	movedE[bID * BLOCK_SIZE_X * BLOCK_SIZE_Y + BLOCK_SIZE_X * ty + tx] = tx - SPXL;
	movedS[bID * BLOCK_SIZE_X * BLOCK_SIZE_Y + BLOCK_SIZE_X * ty + tx] = ty - SPYU;
}




void mexFunction( int nlhs, mxArray *plhs[],
                  int nrhs, const mxArray *prhs[])
{
	float *Im1f, *Im2f, *Sp2f, *Im1_d, *Im2_d, *Sp2_d, *Sp2Val_d;
	double *Im1, *Im2, *Sp2, *boundsX, *boundsY, *Sp2Val;
	double noHits;    
    int i,j, pos, noPels, NC, M, N; 	
    int *xmin, *ymin, *xmax, *ymax, *xmin_d, *ymin_d, *xmax_d, *ymax_d;
    int *movedS, *movedE, *movedS_d, *movedE_d, *bestmovedS, *bestmovedE;
    float *maxScore_d, *maxScoref, *bestScores, *Sp2Valf;	    
    double *maxScoreRes, *movedSRes, *movedERes;

    //double N0, lambda;
	//int flag;
	//float *meanT, *meanTd;
	//flag = mxIsDouble(prhs[0]) ;

    /* Find the dimensions of the data */
    M = mxGetM(prhs[0]);
    N = mxGetN(prhs[0]);
    NC = mxGetM(prhs[2]);
           
    /* Retrieve the input data */
    Im1 = mxGetPr(prhs[0]);
    Im2 = mxGetPr(prhs[1]);
    
    boundsX = mxGetPr(prhs[2]);
    boundsY = mxGetPr(prhs[3]);
    
    Sp2 = mxGetPr(prhs[4]);
    Sp2Val = mxGetPr(prhs[5]);
    noHits = mxGetScalar(prhs[6]);
   
    //N0 = mxGetScalar(prhs[6]);
    //lambda = mxGetScalar(prhs[7]);

	noPels = M*N;
    
    /* Check if the input array is single or double precision */
    Im1f = (float *) mxMalloc(noPels*sizeof(float));
    for (j = 0; j < M*N; j++)
    {
        Im1f[j] = (float) Im1[j];
    }
    Im2f = (float *) mxMalloc(noPels*sizeof(float));
    for (j = 0; j < M*N; j++)
    {
        Im2f[j] = (float) Im2[j];
    }
    Sp2f = (float *) mxMalloc(noPels*sizeof(float));
    for (j = 0; j < M*N; j++)
    {
        Sp2f[j] = (float) Sp2[j];
    }
    Sp2Valf = (float *) mxMalloc(NC*sizeof(float));
	for (j = 0; j < NC; j++)
    {
        Sp2Valf[j] = (float) Sp2Val[j];
    }
    xmin = (int *) mxMalloc(NC*sizeof(int));
	for (j = 0; j < NC; j++)
    {
        xmin[j] = (int) boundsX[j];
    }
    xmax = (int *) mxMalloc(NC*sizeof(int));
	for (j = 0; j < NC; j++)
    {
        xmax[j] = (int) boundsX[NC+j];
    }
	ymin = (int *) mxMalloc(NC*sizeof(int));
	for (j = 0; j < NC; j++)
    {
        ymin[j] = (int) boundsY[j];
    }
	ymax = (int *) mxMalloc(NC*sizeof(int));
	for (j = 0; j < NC; j++)
    {
        ymax[j] = (int) boundsY[NC+j];
    }

   	maxScoref = (float *) mxMalloc(NC*(SPYU+SPYD+1)*(SPXL+SPXR+1)*sizeof(float));
	movedS = (int *) mxMalloc(NC*(SPYU+SPYD+1)*(SPXL+SPXR+1)*sizeof(int));
	movedE = (int *) mxMalloc(NC*(SPYU+SPYD+1)*(SPXL+SPXR+1)*sizeof(int));
	//meanT = (float *) malloc( BLOCK_SIZE_X*BLOCK_SIZE_Y*sizeof(float));

	hipError_t error ;

	error = hipMalloc((void **) &Im1_d, noPels*sizeof(float));
	error = hipMalloc((void **) &Im2_d, noPels*sizeof(float));
	error = hipMalloc((void **) &Sp2_d, noPels*sizeof(float));
	error = hipMalloc((void **) &Sp2Val_d, NC*sizeof(float));
	error = hipMalloc((void **) &xmin_d, NC*sizeof(int));
	error = hipMalloc((void **) &xmax_d, NC*sizeof(int));
	error = hipMalloc((void **) &ymin_d, NC*sizeof(int));
	error = hipMalloc((void **) &ymax_d, NC*sizeof(int));
	error = hipMalloc((void **) &maxScore_d, NC*(SPYU+SPYD+1)*(SPXL+SPXR+1)*sizeof(float));
	error = hipMalloc((void **) &movedS_d, NC*(SPYU+SPYD+1)*(SPXL+SPXR+1)*sizeof(int));
	error = hipMalloc((void **) &movedE_d, NC*(SPYU+SPYD+1)*(SPXL+SPXR+1)*sizeof(int));

    error = hipMemcpy(Im1_d, Im1f, noPels*sizeof(float), hipMemcpyHostToDevice);
	error = hipMemcpy(Im2_d, Im2f, noPels*sizeof(float), hipMemcpyHostToDevice);
	error = hipMemcpy(xmin_d, xmin, NC*sizeof(int), hipMemcpyHostToDevice);
	error = hipMemcpy(xmax_d, xmax, NC*sizeof(int), hipMemcpyHostToDevice);
	error = hipMemcpy(ymin_d, ymin, NC*sizeof(int), hipMemcpyHostToDevice);
	error = hipMemcpy(ymax_d, ymax, NC*sizeof(int), hipMemcpyHostToDevice);
    error = hipMemcpy(Sp2_d, Sp2f, noPels*sizeof(float), hipMemcpyHostToDevice);
    error = hipMemcpy(Sp2Val_d, Sp2Valf, NC*sizeof(float), hipMemcpyHostToDevice);


 	dim3 dimBlock(BLOCK_SIZE_X, BLOCK_SIZE_Y);
 	dim3 dimGrid(NC,1);
 	
	printf("%d\n",sizeof(float));
	printf("Starting GPU...\n");	
	NSSD<<<dimGrid, dimBlock>>>(Im1_d, Im2_d, N, M, xmin_d, xmax_d, ymin_d, ymax_d, maxScore_d, movedS_d, movedE_d, Sp2_d, Sp2Val_d);
	printf("GPU Completed...\n");
	hipMemcpy(maxScoref, maxScore_d, NC*(SPYU+SPYD+1)*(SPXL+SPXR+1)*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(movedS, movedS_d, NC*(SPYU+SPYD+1)*(SPXL+SPXR+1)*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(movedE, movedE_d, NC*(SPYU+SPYD+1)*(SPXL+SPXR+1)*sizeof(int), hipMemcpyDeviceToHost);

	bestScores = (float *) mxMalloc(NC*noHits*sizeof(float));
	bestmovedS = (int *) mxMalloc(NC*noHits*sizeof(int));
	bestmovedE = (int *) mxMalloc(NC*noHits*sizeof(int));
	for (j = 0; j < NC*(SPYU+SPYD+1)*(SPXL+SPXR+1); j+=(SPYU+SPYD+1)*(SPXL+SPXR+1))
    {
		quickSort(&maxScoref[j],(SPYU+SPYD+1)*(SPXL+SPXR+1),&movedS[j],&movedE[j]);
		for (i = 0;i<noHits;i++)
		{
			pos = ( j/((SPYU+SPYD+1)*(SPXL+SPXR+1)) )*noHits+i;
			//sth wrong here
			bestScores[pos] = maxScoref[j+(SPYU+SPYD+1)*(SPXL+SPXR+1)-1-i];
			bestmovedS[pos] = movedS[j+(SPYU+SPYD+1)*(SPXL+SPXR+1)-1-i];
			bestmovedE[pos] = movedE[j+(SPYU+SPYD+1)*(SPXL+SPXR+1)-1-i];
		}
	}

    /* Setup the output */
    plhs[0] = mxCreateDoubleMatrix(noHits,NC,mxREAL);
    maxScoreRes  = mxGetPr(plhs[0]);
    plhs[1] = mxCreateDoubleMatrix(noHits,NC,mxREAL);
    movedSRes  = mxGetPr(plhs[1]);
    plhs[2] = mxCreateDoubleMatrix(noHits,NC,mxREAL);
    movedERes  = mxGetPr(plhs[2]);
	for (j = 0; j < NC*noHits; j++)
    {
        maxScoreRes[j] = (double) bestScores[j];
    }
	for (j = 0; j < NC*noHits; j++)
    {
        movedSRes[j] = (double) bestmovedS[j];
    }
	for (j = 0; j < NC*noHits; j++)
    {
        movedERes[j] = (double) bestmovedE[j];
    }


	hipFree(Im1_d);
	hipFree(Im2_d);
	hipFree(xmin_d);
	hipFree(xmax_d);
	hipFree(ymin_d);
	hipFree(ymax_d);
	hipFree(maxScore_d);
	hipFree(movedS_d);
	hipFree(movedE_d);
	hipFree(Sp2_d);
	hipFree(Sp2Val_d);
	
	mxFree(Im1f);
	mxFree(Im2f);
	mxFree(Sp2f);
	mxFree(Sp2Valf);
	mxFree(xmin);
	mxFree(xmax);
	mxFree(ymin);
	mxFree(ymax);
	mxFree(maxScoref);
	mxFree(movedS);
	mxFree(movedE);
	mxFree(bestScores);
	mxFree(bestmovedS);
	mxFree(bestmovedE);

		
}



